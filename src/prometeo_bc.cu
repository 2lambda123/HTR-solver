#include "hip/hip_runtime.h"
// Copyright (c) "2019, by Stanford University
//               Developer: Mario Di Renzo
//               Affiliation: Center for Turbulence Research, Stanford University
//               URL: https://ctr.stanford.edu
//               Citation: Di Renzo, M., Lin, F., and Urzay, J. (2020).
//                         HTR solver: An open-source exascale-oriented task-based
//                         multi-GPU high-order code for hypersonic aerothermodynamics.
//                         Computer Physics Communications 255, 107262"
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//    * Redistributions of source code must retain the above copyright
//      notice, this list of conditions and the following disclaimer.
//    * Redistributions in binary form must reproduce the above copyright
//      notice, this list of conditions and the following disclaimer in the
//      documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "prometeo_bc.hpp"
#include "prometeo_variables.hpp"
#include "cuda_utils.hpp"

// Declare a constant memory that will hold the Mixture struct (initialized in prometeo_mixture.cu)
extern __device__ __constant__ Mix mix;

//-----------------------------------------------------------------------------
// KERNELS FOR AddRecycleAverageTask
//-----------------------------------------------------------------------------

__global__
void AddRecycleAverageTask_kernel(const AccessorRO<  Vec3, 3> cellWidth,
                                  const AccessorRO<VecNSp, 3> MolarFracs_profile,
                                  const AccessorRO<double, 3> temperature_profile,
                                  const AccessorRO<  Vec3, 3> velocity_profile,
                                  const AccessorSumRD<VecNSp, 1> avg_MolarFracs,
                                  const AccessorSumRD<  Vec3, 1> avg_velocity,
                                  const AccessorSumRD<double, 1> avg_temperature,
                                  const AccessorSumRD<double, 1> avg_rho,
                                  const double Pbc,
                                  const Rect<3> my_bounds,
                                  const coord_t  size_x,
                                  const coord_t  size_y,
                                  const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      AddRecycleAverageTask::collectAverages(cellWidth,
                     MolarFracs_profile, temperature_profile, velocity_profile,
                     avg_MolarFracs, avg_velocity, avg_temperature,
                     avg_rho, Pbc, p, mix);
   }
}

__host__
void AddRecycleAverageTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 4);
   assert(futures.size() == 0);

   // Accessors for cellWidth
   const AccessorRO<  Vec3, 3> acc_cellWidth           (regions[0], FID_cellWidth);

   // Accessors for profile variables
   const AccessorRO<VecNSp, 3> acc_MolarFracs_profile  (regions[0], FID_MolarFracs_profile);
   const AccessorRO<double, 3> acc_temperature_profile (regions[0], FID_temperature_profile);
   const AccessorRO<  Vec3, 3> acc_velocity_profile    (regions[0], FID_velocity_profile);

   // Accessors for averages
   const AccessorSumRD<double, 1> acc_avg_rho          (regions[1], RA_FID_rho,         LEGION_REDOP_SUM_FLOAT64);
   const AccessorSumRD<double, 1> acc_avg_temperature  (regions[1], RA_FID_temperature, LEGION_REDOP_SUM_FLOAT64);
   const AccessorSumRD<VecNSp, 1> acc_avg_MolarFracs   (regions[2], RA_FID_MolarFracs,  REGENT_REDOP_SUM_VECNSP);
   const AccessorSumRD<  Vec3, 1> acc_avg_velocity     (regions[3], RA_FID_velocity,    REGENT_REDOP_SUM_VEC3);

   // Extract execution domain
   Rect<3> r_plane = runtime->get_index_space_domain(ctx, args.plane.get_index_space());

   // Launch the kernel
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_plane);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_plane) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_plane) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_plane) + (TPB_3d.z - 1)) / TPB_3d.z);
   AddRecycleAverageTask_kernel<<<num_blocks_3d, TPB_3d>>>(acc_cellWidth,
                            acc_MolarFracs_profile, acc_temperature_profile, acc_velocity_profile,
                            acc_avg_MolarFracs, acc_avg_velocity, acc_avg_temperature, acc_avg_rho,
                            args.Pbc, r_plane,
                            getSize<Xdir>(r_plane), getSize<Ydir>(r_plane), getSize<Zdir>(r_plane));
}

//-----------------------------------------------------------------------------
// KERNELS FOR SetNSCBC_InflowBC
//-----------------------------------------------------------------------------

template<direction dir>
__global__
void SetNSCBC_InflowBC_kernel(const AccessorRO<VecNEq, 3> Conserved,
                              const AccessorRO<double, 3> SoS,
                              const AccessorRO<VecNSp, 3> MolarFracs_profile,
                              const AccessorRO<double, 3> temperature_profile,
                              const AccessorRO<  Vec3, 3> velocity_profile,
                              const AccessorWO<double, 3> pressure,
                              const AccessorWO<double, 3> temperature,
                              const AccessorWO<VecNSp, 3> MolarFracs,
                              const AccessorWO<  Vec3, 3> velocity,
                              const double Pbc,
                              const Rect<3> my_bounds,
                              const coord_t  size_x,
                              const coord_t  size_y,
                              const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   // Index of normal direction
   constexpr int iN = normalIndex(dir);

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      MolarFracs[p] = MolarFracs_profile[p];
      temperature[p] = temperature_profile[p];
      velocity[p] = velocity_profile[p];
      if (fabs(velocity_profile[p][iN]) >= SoS[p])
         // It is supersonic, everything is imposed by the BC
         pressure[p] = Pbc;
      else
         // Compute pressure from NSCBC conservation equations
         SetNSCBC_InflowBCTask<dir>::setInflowPressure(
                           Conserved, MolarFracs_profile, temperature_profile,
                           pressure, p, mix);
   }
}

template<direction dir>
__host__
void SetNSCBC_InflowBCTask<dir>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 2);
   assert(futures.size() == 0);

   // Accessor for conserved variables
   const AccessorRO<VecNEq, 3> acc_Conserved           (regions[0], FID_Conserved);

   // Accessor for speed of sound
   const AccessorRO<double, 3> acc_SoS                 (regions[0], FID_SoS);

   // Accessors for profile variables
   const AccessorRO<VecNSp, 3> acc_MolarFracs_profile  (regions[0], FID_MolarFracs_profile);
   const AccessorRO<double, 3> acc_temperature_profile (regions[0], FID_temperature_profile);
   const AccessorRO<  Vec3, 3> acc_velocity_profile    (regions[0], FID_velocity_profile);

   // Accessors for primitive variables
   const AccessorWO<double, 3> acc_pressure            (regions[1], FID_pressure);
   const AccessorWO<double, 3> acc_temperature         (regions[1], FID_temperature);
   const AccessorWO<VecNSp, 3> acc_MolarFracs          (regions[1], FID_MolarFracs);
   const AccessorWO<  Vec3, 3> acc_velocity            (regions[1], FID_velocity);

   // Extract execution domain
   Rect<3> r_BC = runtime->get_index_space_domain(ctx,
      runtime->get_logical_subregion_by_color(args.Fluid_BC, 0).get_index_space());

   // Launch the kernel
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_BC);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_BC) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_BC) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_BC) + (TPB_3d.z - 1)) / TPB_3d.z);
   SetNSCBC_InflowBC_kernel<dir><<<num_blocks_3d, TPB_3d>>>(acc_Conserved, acc_SoS,
                        acc_MolarFracs_profile, acc_temperature_profile, acc_velocity_profile,
                        acc_pressure, acc_temperature, acc_MolarFracs, acc_velocity,
                        args.Pbc, r_BC,
                        getSize<Xdir>(r_BC), getSize<Ydir>(r_BC), getSize<Zdir>(r_BC));
}

template void SetNSCBC_InflowBCTask<Xdir>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void SetNSCBC_InflowBCTask<Ydir>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void SetNSCBC_InflowBCTask<Zdir>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

//-----------------------------------------------------------------------------
// KERNELS FOR SetNSCBC_OutflowBC
//-----------------------------------------------------------------------------

__global__
void SetNSCBC_OutflowBC_kernel(const AccessorRO<VecNEq, 3> Conserved,
                               const AccessorRW<double, 3> temperature,
                               const AccessorWO<double, 3> pressure,
                               const AccessorWO<VecNSp, 3> MolarFracs,
                               const AccessorWO<  Vec3, 3> velocity,
                               const Rect<3> my_bounds,
                               const coord_t  size_x,
                               const coord_t  size_y,
                               const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      UpdatePrimitiveFromConservedTask::UpdatePrimitive(
                     Conserved, temperature, pressure,
                     MolarFracs, velocity,
                     p, mix);
   }
}


__host__
void SetNSCBC_OutflowBCTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 2);
   assert(futures.size() == 0);

   // Accessors for conserved variables
   const AccessorRO<VecNEq, 3> acc_Conserved        (regions[0], FID_Conserved);

   // Accessors for temperature variables
   const AccessorRW<double, 3> acc_temperature      (regions[1], FID_temperature);

   // Accessors for primitive variables
   const AccessorWO<double, 3> acc_pressure         (regions[1], FID_pressure);
   const AccessorWO<VecNSp, 3> acc_MolarFracs       (regions[1], FID_MolarFracs);
   const AccessorWO<  Vec3, 3> acc_velocity         (regions[1], FID_velocity);

   // Extract execution domain
   Rect<3> r_BC = runtime->get_index_space_domain(ctx,
      runtime->get_logical_subregion_by_color(args.Fluid_BC, 0).get_index_space());

   // Launch the kernel
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_BC);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_BC) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_BC) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_BC) + (TPB_3d.z - 1)) / TPB_3d.z);
   SetNSCBC_OutflowBC_kernel<<<num_blocks_3d, TPB_3d>>>(
                        acc_Conserved, acc_temperature, acc_pressure,
                        acc_MolarFracs, acc_velocity, r_BC,
                        getSize<Xdir>(r_BC), getSize<Ydir>(r_BC), getSize<Zdir>(r_BC));
}

//-----------------------------------------------------------------------------
// KERNELS FOR SetIncomingShockBCTask
//-----------------------------------------------------------------------------

__global__
void SetIncomingShockBC_kernel(const AccessorRO<VecNEq, 3> Conserved,
                               const AccessorRO<double, 3> SoS,
                               const AccessorWO<double, 3> temperature,
                               const AccessorWO<double, 3> pressure,
                               const AccessorWO<VecNSp, 3> MolarFracs,
                               const AccessorWO<  Vec3, 3> velocity,
                               const Vec3 velocity0,
                               const double temperature0,
                               const double pressure0,
                               const Vec3 velocity1,
                               const double temperature1,
                               const double pressure1,
                               const VecNSp MolarFracs0,
                               const double MixW,
                               const int iShock,
                               const Rect<3> my_bounds,
                               const coord_t  size_x,
                               const coord_t  size_y,
                               const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      if (p.x < iShock) {
         // Set to upstream values
         MolarFracs[p]  = MolarFracs0;
         velocity[p]    = velocity0;
         temperature[p] = temperature0;
         pressure[p]    = pressure0;

      } else if (p.x > iShock) {
         // Treat this point as an NSCBCInflow
         MolarFracs[p]  = MolarFracs0;
         velocity[p]    = velocity1;
         temperature[p] = temperature1;
         if (fabs(velocity1[1]) >= SoS[p])
            // It is supersonic, everything is imposed by the BC
            pressure[p] = pressure1;
         else
            // Compute pressure from NSCBC conservation equations
            pressure[p] = SetIncomingShockBCTask::setPressure(Conserved, temperature1, MixW, p, mix);

      } else {
         // Set to downstream values
         MolarFracs[p]  = MolarFracs0;
         velocity[p]    = velocity1;
         temperature[p] = temperature1;
         pressure[p]    = pressure1;

      }
   }
}

__host__
void SetIncomingShockBCTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 2);
   assert(futures.size() == 0);

   // Accessor for conserved variables
   const AccessorRO<VecNEq, 3> acc_Conserved           (regions[0], FID_Conserved);

   // Accessor for speed of sound
   const AccessorRO<double, 3> acc_SoS                 (regions[0], FID_SoS);

   // Accessors for primitive variables
   const AccessorWO<double, 3> acc_pressure            (regions[1], FID_pressure);
   const AccessorWO<double, 3> acc_temperature         (regions[1], FID_temperature);
   const AccessorWO<VecNSp, 3> acc_MolarFracs          (regions[1], FID_MolarFracs);
   const AccessorWO<  Vec3, 3> acc_velocity            (regions[1], FID_velocity);

   // Extract execution domain
   Rect<3> r_BC = runtime->get_index_space_domain(ctx,
      runtime->get_logical_subregion_by_color(args.Fluid_BC, 0).get_index_space());

   // Precompute the mixture averaged molecular weight
   VecNSp MolarFracs(args.params.MolarFracs);
   const double MixW = args.mix.GetMolarWeightFromXi(MolarFracs);

   // Launch the kernel
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_BC);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_BC) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_BC) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_BC) + (TPB_3d.z - 1)) / TPB_3d.z);
   SetIncomingShockBC_kernel<<<num_blocks_3d, TPB_3d>>>(
                        acc_Conserved, acc_SoS,
                        acc_temperature, acc_pressure, acc_MolarFracs, acc_velocity,
                        Vec3(args.params.velocity0), args.params.temperature0, args.params.pressure0,
                        Vec3(args.params.velocity1), args.params.temperature1, args.params.pressure1,
                        MolarFracs, MixW, args.params.iShock,
                        r_BC, getSize<Xdir>(r_BC), getSize<Ydir>(r_BC), getSize<Zdir>(r_BC));
}

//-----------------------------------------------------------------------------
// KERNELS FOR SetRecycleRescalingBCTask
//-----------------------------------------------------------------------------

#ifdef BOUNDS_CHECKS
   // See Legion issue #879 for more info
   #warning "CUDA variant of RecycleRescalingBC is not available with BOUNDS_CHECKS"
#else
__global__
void SetRecycleRescalingBC_kernel(const AccessorRO<  Vec3, 3> centerCoordinates,
                                  const AccessorRO<VecNEq, 3> Conserved,
                                  const AccessorRO<double, 3> SoS,
                                  const AccessorWO<double, 3> temperature,
                                  const AccessorWO<double, 3> pressure,
                                  const AccessorWO<VecNSp, 3> MolarFracs,
                                  const AccessorWO<  Vec3, 3> velocity,
                                  const AccessorRO<double, 3> temperature_recycle,
                                  const AccessorRO<  Vec3, 3> velocity_recycle,
                                  const AccessorRO<VecNSp, 3> MolarFracs_recycle,
                                  const AccessorRO<double, 3> temperature_profile,
                                  const AccessorRO<  Vec3, 3> velocity_profile,
                                  const AccessorRO<VecNSp, 3> MolarFracs_profile,
                                  const AccessorRO<double, 1> avg_y,
                                  const AccessorRO< float, 1> FI_xloc,
                                  const AccessorRO< float, 1> FI_iloc,
                                  const FastInterpData FIdata,
                                  const double Pbc,
                                  const double yInnFact,
                                  const double yOutFact,
                                  const double uInnFact,
                                  const double uOutFact,
                                  const double idelta99Inl,
                                  const Rect<3> my_bounds,
                                  const coord_t  size_x,
                                  const coord_t  size_y,
                                  const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);

      // Compute the rescaled primitive quantities
      double temperatureR; Vec3 velocityR; VecNSp MolarFracsR;
      SetRecycleRescalingBCTask::GetRescaled(
                  temperatureR, velocityR, MolarFracsR, centerCoordinates,
                  temperature_recycle, velocity_recycle, MolarFracs_recycle,
                  temperature_profile, velocity_profile, MolarFracs_profile,
                  avg_y, FI_xloc, FI_iloc, FIdata, p,
                  yInnFact, yOutFact, uInnFact, uOutFact, idelta99Inl);

      MolarFracs[p] = MolarFracsR;
      temperature[p] = temperatureR;
      velocity[p] = velocityR;
      if (fabs(velocityR[0]) >= SoS[p])
         // It is supersonic, everything is imposed by the BC
         pressure[p] = Pbc;
      else
         // Compute pressure from NSCBC conservation equations
         pressure[p] = SetRecycleRescalingBCTask::setPressure(Conserved, temperatureR, MolarFracsR, p, mix);
   }
}
#endif

__host__
void SetRecycleRescalingBCTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
#ifdef BOUNDS_CHECKS
   // See Legion issue #879 for more info
   #warning "CUDA variant of RecycleRescalingBC is not available with BOUNDS_CHECKS"
#else
   assert(regions.size() == 5);
   assert(futures.size() == 1);

   // Accessor for speed of sound
   const AccessorRO<  Vec3, 3> acc_centerCoordinates   (regions[0], FID_centerCoordinates);

   // Accessor for conserved variables
   const AccessorRO<VecNEq, 3> acc_Conserved           (regions[0], FID_Conserved);

   // Accessor for speed of sound
   const AccessorRO<double, 3> acc_SoS                 (regions[0], FID_SoS);

   // Accessors for profile variables
   const AccessorRO<VecNSp, 3> acc_MolarFracs_profile  (regions[0], FID_MolarFracs_profile);
   const AccessorRO<double, 3> acc_temperature_profile (regions[0], FID_temperature_profile);
   const AccessorRO<  Vec3, 3> acc_velocity_profile    (regions[0], FID_velocity_profile);

   // Accessors for primitive variables
   const AccessorWO<double, 3> acc_pressure            (regions[1], FID_pressure);
   const AccessorWO<double, 3> acc_temperature         (regions[1], FID_temperature);
   const AccessorWO<VecNSp, 3> acc_MolarFracs          (regions[1], FID_MolarFracs);
   const AccessorWO<  Vec3, 3> acc_velocity            (regions[1], FID_velocity);

   // Accessors for avg wall-normal coordinate
   const AccessorRO<double, 1> acc_avg_y               (regions[2], RA_FID_y);

   // Accessors for recycle plane variables
   const AccessorRO<VecNSp, 3> acc_MolarFracs_recycle  (regions[3], FID_MolarFracs_recycle);
   const AccessorRO<double, 3> acc_temperature_recycle (regions[3], FID_temperature_recycle);
   const AccessorRO<  Vec3, 3> acc_velocity_recycle    (regions[3], FID_velocity_recycle);

   // Accessors for fast interpolation region
   const AccessorRO< float, 1> acc_FI_xloc             (regions[4], FI_FID_xloc);
   const AccessorRO< float, 1> acc_FI_iloc             (regions[4], FI_FID_iloc);

   // Extract execution domain
   Rect<3> r_BC = runtime->get_index_space_domain(ctx,
      runtime->get_logical_subregion_by_color(args.Fluid_BC, 0).get_index_space());

   // Compute rescaling coefficients
   const RescalingDataType RdataRe = futures[0].get_result<RescalingDataType>();
   const double yInnFact = RdataRe.deltaNu  /args.RdataIn.deltaNu;
   const double yOutFact = RdataRe.delta99VD/args.RdataIn.delta99VD;
   const double uInnFact = args.RdataIn.uTau/RdataRe.uTau;
   const double uOutFact = uInnFact*sqrt(args.RdataIn.rhow/RdataRe.rhow);

   const double idelta99Inl = 1.0/args.RdataIn.delta99VD;

   // Launch the kernel
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_BC);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_BC) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_BC) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_BC) + (TPB_3d.z - 1)) / TPB_3d.z);
   SetRecycleRescalingBC_kernel<<<num_blocks_3d, TPB_3d>>>(
                        acc_centerCoordinates, acc_Conserved, acc_SoS,
                        acc_temperature, acc_pressure, acc_MolarFracs, acc_velocity,
                        acc_temperature_recycle, acc_velocity_recycle, acc_MolarFracs_recycle,
                        acc_temperature_profile, acc_velocity_profile, acc_MolarFracs_profile,
                        acc_avg_y, acc_FI_xloc, acc_FI_iloc, args.FIdata, args.Pbc,
                        yInnFact, yOutFact, uInnFact, uOutFact, idelta99Inl,
                        r_BC, getSize<Xdir>(r_BC), getSize<Ydir>(r_BC), getSize<Zdir>(r_BC));
#endif
}

#if (defined(ELECTRIC_FIELD) && (nIons > 0))
//-----------------------------------------------------------------------------
// KERNELS FOR CorrectIonsBCTask
//-----------------------------------------------------------------------------

template<direction dir, side s>
__global__
void CorrectIonsBC_kernel(const AccessorRO<double, 3> ePot,
                          const AccessorRW<VecNSp, 3> MolarFracs,
                          const Rect<3> my_bounds,
                          const coord_t  size_x,
                          const coord_t  size_y,
                          const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      const Point<3> pInt = getPIntBC<dir, s>(p);
      const double dPhi = ePot[pInt] - ePot[p];
      __UNROLL__
      for (int i = 0; i < nIons; i++) {
         int ind = mix.ions[i];
         if (mix.GetSpeciesChargeNumber(ind)*dPhi > 0)
            // the ion is flowing into the BC
            MolarFracs[p][ind] = MolarFracs[pInt][ind];
         else
            // the ion is repelled by the BC
            MolarFracs[p][ind] = 1e-60;
      }
   }
}

template<direction dir, side s>
__host__
void CorrectIonsBCTask<dir, s>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 2);
   assert(futures.size() == 0);

   // Accessor for electric potential
   const AccessorRO<double, 3> acc_ePot       (regions[0], FID_electricPotential);

   // Accessors for primitive variables
   const AccessorRW<VecNSp, 3> acc_MolarFracs (regions[1], FID_MolarFracs);

   // Extract execution domain
   Rect<3> r_BC = runtime->get_index_space_domain(ctx,
      runtime->get_logical_subregion_by_color(args.Fluid_BC, 0).get_index_space());

   // Launch the kernel
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_BC);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_BC) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_BC) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_BC) + (TPB_3d.z - 1)) / TPB_3d.z);
   CorrectIonsBC_kernel<dir, s><<<num_blocks_3d, TPB_3d>>>(
                        acc_ePot, acc_MolarFracs,
                        r_BC, getSize<Xdir>(r_BC), getSize<Ydir>(r_BC), getSize<Zdir>(r_BC));
};

template void CorrectIonsBCTask<Xdir, Minus>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void CorrectIonsBCTask<Xdir, Plus >::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void CorrectIonsBCTask<Ydir, Minus>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void CorrectIonsBCTask<Ydir, Plus >::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void CorrectIonsBCTask<Zdir, Minus>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void CorrectIonsBCTask<Zdir, Plus >::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);
#endif

